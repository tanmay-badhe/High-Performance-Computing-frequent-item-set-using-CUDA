//Technique 1
//Count array act as a multiple bucket set
//frequent-items-using-CUDA

#include<iostream>
#include<hip/hip_runtime.h>

#include<stdio.h>
#include <stdlib.h>
#include<time.h>
#include<fstream>
using namespace std;

__global__ void addKernel(int *a,int *count_d,int *nOfItemSet_d)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  int val=a[blockIdx.x*blockDim.x+threadIdx.x];
  atomicAdd(&count_d[1000*(int(i/5000))+val],1);		//right
  //count_d[threadIdx.x]=a[threadIdx.x];
  if(i<1000){
	for(int j=1;j< *nOfItemSet_d;i++){
		count_d[i]=count_d[i]+count_d[j*1000+i];
	}
  }
}

int main(){
int n;
cout<<"enter number of transaction";
cin>>n;
ifstream in("out.txt");
int *a_d,*a_h,*count_d,*count_h,*nOfItemSet_d,*nOfItemSet_h;
int size=n*sizeof(int);
int size1=1000*sizeof(int);

a_h=(int*)malloc(n*sizeof(int));
count_h=(int*)malloc(1000*sizeof(int));
nOfItemSet_h=(int*)malloc(sizeof(int));

for(int i=0;i<n;i++)
{
	in>>a_h[i];
}

for(int i=0;i<1000;i++)
	count_h[i]=0;
*nOfItemSet_h=((n-1)/5000)+1;

hipMalloc((void**)&a_d,size);
hipMalloc((void**)&count_d,size1);
hipMalloc((void**)&nOfItemSet_d,sizeof(int));
hipMemcpy(a_d,a_h,size,hipMemcpyHostToDevice);
hipMemcpy(count_d,count_h,size1,hipMemcpyHostToDevice);
hipMemcpy(nOfItemSet_d,nOfItemSet_h,sizeof(int),hipMemcpyHostToDevice);

addKernel<<<((n-1)/256)+1,256>>>(a_d,count_d,nOfItemSet_d);

hipMemcpy(count_h,count_d,size1,hipMemcpyDeviceToHost);
hipFree(a_d);
hipFree(count_d);
for(int i=0;i<1000;i++)
	cout<<i<<" "<<count_h[i]<<endl;
//cout<<"time taken: "<<(double) (end-start) / CLOCKS_PER_SEC * 1000.0;
return 0;

}

