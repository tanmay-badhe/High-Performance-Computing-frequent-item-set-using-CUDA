#include<iostream>
#include<hip/hip_runtime.h>

#include<stdio.h>
#include <stdlib.h>
#include<time.h>
#include<fstream>
using namespace std;

__global__ void addKernel(int *a,int *count_d)
{
  __shared__ int c[1000];
  int i=threadIdx.x;
  int val=a[blockIdx.x*blockDim.x+threadIdx.x];
  if(i<250){
     c[i]=0;
     c[i+250]=0;
     c[i+500]=0;
     c[i+750]=0;
 }
  __syncthreads();
  atomicAdd(&c[val],1);
  __syncthreads();
  if(i<250){
	atomicAdd(&count_d[i],c[i]);
	atomicAdd(&count_d[i+250],c[i+250]);
	atomicAdd(&count_d[i+500],c[i+500]);
	atomicAdd(&count_d[i+750],c[i+750]);
  }
  
}

int main(){
int n;
cout<<"enter number of transaction";
cin>>n;
ifstream in("out.txt");
int *a_d,*a_h,*count_d,*count_h;
int size=n*sizeof(int);
int size1=1000*sizeof(int);
a_h=(int*)malloc(n*sizeof(int));
count_h=(int*)malloc(1000*sizeof(int));
for(int i=0;i<n;i++)
{
	in>>a_h[i];
}
for(int i=0;i<1000;i++)
	count_h[i]=0;
hipMalloc((void**)&a_d,size);
hipMalloc((void**)&count_d,size1);
hipMemcpy(a_d,a_h,size,hipMemcpyHostToDevice);
hipMemcpy(count_d,count_h,size1,hipMemcpyHostToDevice);
addKernel<<<((n-1)/256)+1,256>>>(a_d,count_d);
hipMemcpy(count_h,count_d,size1,hipMemcpyDeviceToHost);
hipFree(a_d);
hipFree(count_d);
for(int i=0;i<1000;i++)
	cout<<i<<" "<<count_h[i]<<endl;
return 0;

}

