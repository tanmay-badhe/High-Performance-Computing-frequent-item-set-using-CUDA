#include "hip/hip_runtime.h"
//Technique 1
//Count array act as a bucket
//frequent-items-using-CUDA
#include<iostream>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<stdio.h>
#include <stdlib.h>
#include<time.h>
#include<fstream>
using namespace std;

__global__ void addKernel(int *a,int *count_d)
{
  int idx=blockIdx.x*blockDim.x+threadIdx.x
  int i=a[idx];
  atomicAdd(&count_d[i],1);		
}

int main(){
int n;
cout<<"enter number of transaction";
cin>>n;
ifstream in("out.txt");
int *a_d,*a_h,*count_d,*count_h;
int size=n*sizeof(int);
int size1=1000*sizeof(int);
a_h=(int*)malloc(n*sizeof(int));
count_h=(int*)malloc(1000*sizeof(int));
for(int i=0;i<n;i++)
{
	in>>a_h[i];
}
for(int i=0;i<1000;i++)
	count_h[i]=0;
hipMalloc((void**)&a_d,size);
hipMalloc((void**)&count_d,size1);
hipMemcpy(a_d,a_h,size,hipMemcpyHostToDevice);
hipMemcpy(count_d,count_h,size1,hipMemcpyHostToDevice);
clock_t start = clock();
addKernel<<<((n-1)/256)+1,256>>>(a_d,count_d);
clock_t end = clock();
hipMemcpy(count_h,count_d,size1,hipMemcpyDeviceToHost);
hipFree(a_d);
hipFree(count_d);
for(int i=0;i<1000;i++)
	cout<<i<<" "<<count_h[i]<<endl;
cout<<"time taken: "<<(double) (end-start) / CLOCKS_PER_SEC * 1000.0;
return 0;

}

